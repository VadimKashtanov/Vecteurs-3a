#include "hip/hip_runtime.h"
/*	Ceci est un fichier a integrer de maniere brute avec tous les templates avec #include 	*/
/*				Pas besoin de tete pour ces templates, mais c'est possible					*/
/*							C'est juste du copier-coller									*/
/*							Ca sera comme des fonctions static 								*/

#include "etc.cuh"

__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ static float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

static __device__ float cuda_signe(float x) {
	//return (x>=0 ? 1:-1);
	if (x >= 0) return 1;
	else return -1;
}


//  - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -

template <typename T>
static T max_lst(T * l, uint I) {
	T _max  = l[0];
	FOR(1, i, I) if (_max < l[i]) _max = l[i];
	return _max;
};

//  - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -

template <typename T>
static T * alloc(uint A)
{
	return (T*)malloc(sizeof(T) * A);
}

template <typename T>
static T * liste(uint A, ...) {
	T val;
	va_list vl;
	va_start(vl,A);
	T * ret = alloc<T>(A);
	for (uint i=1; i < A; i++) ret[i] = va_arg(vl, T);
	va_end(vl);
	return ret;
};

static char * join(uint A, ...) {
	va_list vl;
	va_start(vl,A);
	char * liste[A];
	FOR(0, i, A) {
		liste[i] = va_arg(vl, char*);
	}
	va_end(vl);
	//
	uint taille = 1;
	FOR(0, i, A) taille += strlen(liste[i]);
	char * s = (char*)malloc(taille);
	uint depart = 0;
	FOR(0, i, A) {
		memcpy(s + depart, liste[i], strlen(liste[i]));
		depart += strlen(liste[i]);
	}
	s[taille-1] = '\0';
	//
	puts(s);
	return s;
}

template <typename T>
static T * copier(T * a, uint A)
{
	T * r = alloc<T>(A);
	memcpy(r, a, sizeof(T) * A);
	return r;
}

static float * lst_rnd(uint A, float a, float b)
{
	float * ret = alloc<float>(A);
	FOR(0, i, A) ret[i] = a + rnd()*(b-a);
	return ret;
}

template <typename T>
static T * zero(uint A)
{
	return (T*)calloc(A, sizeof(T));
}

//  - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -

template <typename T>
static T*   lire(FILE * fp,        uint A)
{
	T * ret = alloc<T>(A);
	FREAD(ret, sizeof(T), A, fp);
	return ret;
};

template <typename T>
static T   lire_un(FILE * fp)
{
	T ret;
	FREAD(&ret, sizeof(T), 1, fp);
	return ret;
};

template <typename T>
static void ecrire(FILE * fp, T * l, uint A)
{
	FWRITE(l, sizeof(T), A, fp);
};


template <typename T>
static void ecrire_un(FILE * fp, T a)
{
	FWRITE(&a, sizeof(T), 1, fp);
};

//  - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -

template <typename T> 
static void cudafree(T * l)
{
	CONTROLE_CUDA(hipFree(l));
}

template <typename T> 
static T *     cudalloc(                uint A)
{
	T * ret;
	CONTROLE_CUDA(hipMalloc((void**)&ret, sizeof(T)*A));
	CONTROLE_CUDA(hipMemset(ret, 0, sizeof(T) * A));
	return ret;
}

template <typename T>
static T * cpu_vers_gpu(T * lst,        uint A)
{
	T * ret__d = cudalloc<T>(A);
	CONTROLE_CUDA(hipMemcpy(ret__d, lst, sizeof(T)*A, hipMemcpyHostToDevice));
	return ret__d;
}

template <typename T>
static T * gpu_vers_cpu(T * lst__d,     uint A)
{
	T * ret = alloc<T>(A);
	CONTROLE_CUDA(hipMemcpy(ret, lst__d, sizeof(T)*A, hipMemcpyDeviceToHost));
	return ret;
}

//	-------------------------------------------------------------------

static void   cudaplume(float * lst__d, uint A)
{
	float * r = gpu_vers_cpu<float>(lst__d, A);
	FOR(0, i, A) printf("(%i)%f, ", i, r[i]);
	printf("\n");
	free(r);
}

static void mat_plume(float * lst, uint X, uint Y) {
	printf("   ");
	FOR(0, x, X) printf("     %03i  ", x);
	printf("\n");
	FOR(0, y, Y) {
		printf("%03i| ", y);
		FOR(0, x, X) printf("%+f ", lst[y*X+x]);
		printf("\n");
	}
}

PAS_OPTIMISER()
static void comparer_lst(float * l0, float * l1, uint A, float profondeure) {
	FOR(0, i, A)
	{
		if (fabs(l0[i]-l1[i]) < profondeure) {
			printf("%03i| \033[92m%+f  ~= %+f\033[0m\n", i, l0[i], l1[i]);
		} else {
			printf("%03i| \033[91m%+f =/= %+f\033[0m \033[93m(dist=\033[91m%+f\033[93m)\033[0m\n", i, l0[i], l1[i], fabs(l0[i]-l1[i]));
		}
	};
};

static void comparer_lst_2d(
	float * l0, float * l1,
	uint X,
	uint Y, char * ynom,
	float profondeure)
{
	FOR(0, y, Y)
	{
		printf(" ### %s = %i ###\n", ynom, y);
		FOR(0, x, X)
		{
			uint i = y*X+x;
			if (fabs(l0[i]-l1[i]) < profondeure) {
				printf("%03i|\033[1m%03i\033[0m| \033[92m%+f  ~= %+f\033[0m\n",
					i, x, l0[i], l1[i]);
			} else {
				printf("%03i|\033[1m%03i\033[0m| \033[91m%+f =/= %+f\033[0m \033[93m(dist=\033[91m%+f\033[93m)\033[0m\n",
					i, x, l0[i], l1[i], fabs(l0[i]-l1[i]));
			}
		}
	}
};

static uint egales_lst(float * l0, float * l1, uint A, float profondeure) {
	FOR(0, i, A) {
		if (fabs(l0[i]-l1[i]) > profondeure)
			return 0;
	}
	return 1;
}

//	=============

static float * de_a(float de, float a, uint A) {
	float * ret = alloc<float>(A);
	FOR(0, i, A) ret[i] = de + (a-de)/(A-1)*i;
	return ret;
};

//	============================================================================
//	============================= Kerd Cuda ====================================
//	============================================================================

template <typename T>
__global__
static void kerd_liste_inis(T * l, T elm, uint I)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < I) {
		l[thx] = elm;
	};
};