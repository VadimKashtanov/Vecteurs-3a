#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__f_btcusdt(
	float * somme_score,
	float * y, float * p1p0,
	uint * ts__d,
	uint Y,
	uint T)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (t < GRAND_T) {
		//
		float s=0;
		//
		FOR(0, mega_t, MEGA_T) {
			uint ty        = t_MODE(t, mega_t);
			uint t_btcusdt = ts__d[t] + mega_t;
			assert(t_btcusdt < T);
			//
			float _y = y[ty*Y + 0];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*1 + mega_t];
			//
			s += S(_y, _p1p0);
		}
		//
		atomicAdd(&somme_score[0], s);
	}
};

float f_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint Y = btcusdt->Y;
	//
	//
	float * somme__d = cudalloc<float>(1);
	k__f_btcusdt<<<dim3(KERD(GRAND_T, 16)), dim3(16)>>>(
		somme__d,
		y__d, btcusdt->sorties__d,
		ts__d,
		Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
	//
	//
	float * somme = gpu_vers_cpu<float>(somme__d, 1);
	//
	float score = somme[0] / ((float)(GRAND_T * MEGA_T));
	//
	//
	cudafree<float>(somme__d   );
	    free       (somme      );
	//
	return score;
};