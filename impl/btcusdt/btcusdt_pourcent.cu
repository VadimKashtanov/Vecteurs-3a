#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__pourcent_btcusdt(
	float * somme, float * potentiel,
	float * y, float * p1p0,
	float coef_puissance,
	uint * ts__d,
	uint Y)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (t < GRAND_T && mega_t < MEGA_T) {
		uint ty        = t_MODE(t, mega_t);
		uint t_btcusdt = ts__d[t] + mega_t;
		//
		uint a_t_il_predit = (sng(p1p0[t_btcusdt]) == sng(y[ty*Y + 0]));
		//
		float _____somme = powf(fabs(p1p0[t_btcusdt]), coef_puissance) * a_t_il_predit;
		float _potentiel = powf(fabs(p1p0[t_btcusdt]), coef_puissance) * true         ;
		//
		atomicAdd(&somme    [0], _____somme);
		atomicAdd(&potentiel[0], _potentiel);
	}
};

float *  pourcent_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d, float coef_puissance) {
	uint Y = btcusdt->Y;
	//
	float *     somme__d = cudalloc<float>(1);
	float * potentiel__d = cudalloc<float>(1);
	//
	k__pourcent_btcusdt<<<dim3(KERD(GRAND_T, 16), KERD(MEGA_T, 8)), dim3(16,8)>>>(
		somme__d, potentiel__d,
		y__d, btcusdt->sorties__d,
		coef_puissance,
		ts__d,
		Y
	);
	ATTENDRE_CUDA();
	//
	float * somme     = gpu_vers_cpu<float>(    somme__d, 1);
	float * potentiel = gpu_vers_cpu<float>(potentiel__d, 1);
	//
	float * ret = alloc<float>(1);
	FOR(0, p, 1) ret[p] = somme[p] / potentiel[p];
	//
	cudafree<float>(    somme__d);
	cudafree<float>(potentiel__d);
	    free(           somme   );
	    free(       potentiel   );
	//
	return ret;
};
