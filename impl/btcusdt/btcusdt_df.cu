#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__df_btcusdt(
	float * y, float * p1p0, float * dy,
	uint * ts__d,
	uint Y,
	uint T)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (t < GRAND_T) {
		FOR(0, mega_t, MEGA_T) {
			uint ty        = t_MODE(t, mega_t);
			uint t_btcusdt = ts__d[t] + mega_t;
			assert(t_btcusdt < T);
			//
			//
			float _y = y[ty*Y + 0];
			assert(_y >= -1 && _y <= +1);
			//
			float _p1p0 = p1p0[t_btcusdt*1 + mega_t];
			//
			float _ds = dS(_y, _p1p0);
			dy[ty*Y + 0] = _ds / ((float)(GRAND_T * MEGA_T));
		};
	};
};

void df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint Y = btcusdt->Y;
	//
	//
	k__df_btcusdt<<<dim3(KERD(GRAND_T, 16)), dim3(16)>>>(
		y__d, btcusdt->sorties__d, dy__d,
		ts__d,
		Y,
		btcusdt->T
	);
	ATTENDRE_CUDA();
};