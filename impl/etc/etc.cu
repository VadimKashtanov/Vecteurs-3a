#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../../impl_template/tmpl_etc.cu"

FILE * FOPEN(char * fichier, char * mode) {
	FILE * fp = fopen(fichier, mode);
	if (fp == 0) {
		ERR("%s n'existe pas", fichier);
	}
	return fp;
};

float rnd()
{
#define PROFONDEURE 100000
	return (float)(rand() % PROFONDEURE) / (float)PROFONDEURE;
}

float poid_1_1() {
	float amplitude = 1;//tanh(8*rnd());//powf(2.0, rnd())-1.0;
	float vecteur   = 2*rnd()-1;
	//
	return amplitude * vecteur;
};

float signe(float x)
{
	return (x>=0 ? 1:-1);
};

char * scientifique(uint nb) {
	float x = (float)nb;
	uint dim = 0;
	uint s[100];
	while (x >= 1.0) {
		s[dim] = (uint)(x - (float)((uint)(x/10.0)*10.0));
		x /= 10.0;
		dim++;
	}
	//
	char str[100];
	//
	uint pos_dans = 0;
	FOR(0, i, dim) {
		if (i % 3 == 0 && i != 0) {
			str[pos_dans] = '\'';
			pos_dans++;
		};
		str[pos_dans] = s[i] + '0';
		pos_dans++;
	}
	//
	char * inverse = (char*)malloc(pos_dans+1);
	FOR(0, i, pos_dans) inverse[i] = str[pos_dans-1-i];
	//
	inverse[pos_dans] = '\0';
	//
	return inverse;
};

double secondes()
{
	struct timespec now;
	timespec_get(&now, TIME_UTC);
	return 1000.0*(((int64_t) now.tv_sec) * 1000 + ((int64_t) now.tv_nsec) / 1000000);
};

PAS_OPTIMISER()
void titre(char * str) {
	printf("\033[93m=========\033[0m %s \033[93m=========\033[0m\n", str);
};