#include "hip/hip_runtime.h"
#include "opti.cuh"

__global__ static void kerd_sgd(
	uint t,
	float * p, float * dp,
	//
	float alpha,
	uint POIDS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx];
		//
		float ch  = alpha * _grad;
		float reg = alpha * L2_regularisation * p[thx];
		//
		p[thx] -= (ch + reg);
	}
};

void sgd(
	Mdl_t   * mdl,
	float *** hist,
	uint         i,
	float    alpha,
	uint         t
) {
	FOR(0, i, mdl->insts) {
		Inst_t * inst = mdl->inst[i];
		//
		if (inst->P != 0) {
			kerd_sgd<<<dim3(KERD(mdl->inst[i]->P, 256)),dim3(256)>>>(
				t,
				inst->p__d, inst->dp__d,
				//
				alpha,
				inst->P
			);
		}
	}
	ATTENDRE_CUDA();
};