#include "hip/hip_runtime.h"
#include "opti.cuh"

#define rms 0.9

__global__ static void kerd_rmsprop(
	uint t,
	float * p, float * dp,
	float * s,
	float alpha,
	uint POIDS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx];
		//
		float _s = rms*s[thx] + (1-rms)*_grad*_grad;
		//
		s[thx] = _s;
		//
		float ch  = alpha * _grad / (sqrtf(_s + 1e-8));
		float reg = alpha * L2_regularisation * p[thx];
		//
		p[thx] -= (ch + reg);
	}
};

void rmsprop(
	Mdl_t   * mdl,
	float *** hist,
	uint         i,
	float    alpha,
	uint         t
) {
	FOR(0, i, mdl->insts) {
		Inst_t * inst = mdl->inst[i];
		//
		if (inst->P != 0) {
			kerd_rmsprop<<<dim3(KERD(mdl->inst[i]->P, 256)),dim3(256)>>>(
				t,
				inst->p__d, inst->dp__d,
				hist[0][i],
				alpha,
				inst->P
			);
		}
	}
	ATTENDRE_CUDA();
};