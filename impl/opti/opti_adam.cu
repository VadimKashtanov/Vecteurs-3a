#include "hip/hip_runtime.h"
#include "opti.cuh"

#define adam_beta1 0.9 
#define adam_beta2 0.98

__global__ static void kerd_adam(
	uint t,
	float * p, float * dp,
	float * v, float * s,
	float alpha,
	uint POIDS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx];
		//
		float _v = adam_beta1*v[thx] + (1-adam_beta1)*_grad;
		float _s = adam_beta2*s[thx] + (1-adam_beta2)*_grad*_grad;
		//
		v[thx] = _v;
		s[thx] = _s;
		//
		float corr_v = _v / (1.0 - powf(adam_beta1,1+t));
		float corr_s = _s / (1.0 - powf(adam_beta2,1+t));
		//
		float ch  = alpha * corr_v / (sqrtf(corr_s) + 1e-8);
		float reg = alpha * L2_regularisation * p[thx];
		//
		p[thx] -= (ch + reg);
	}
};

void adam(
	Mdl_t   * mdl,
	float *** hist,
	uint         i,
	float    alpha,
	uint         t
) {
	FOR(0, i, mdl->insts) {
		Inst_t * inst = mdl->inst[i];
		//
		if (inst->P != 0) {
			kerd_adam<<<dim3(KERD(mdl->inst[i]->P, 256)),dim3(256)>>>(
				t,
				inst->p__d, inst->dp__d,
				hist[0][i], hist[1][i],
				alpha,
				inst->P
			);
		}
	}
	ATTENDRE_CUDA();
};