#include "hip/hip_runtime.h"
#include "opti.cuh"

#define moment_p 0.9

__global__ static void kerd_moment(
	uint t,
	float * p, float * dp,
	float * m,
	float alpha,
	uint POIDS)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx];
		//
		float _m = (1-moment_p)*m[thx] + moment_p*_grad;
		m[thx] = _m;
		//
		float ch  = alpha * _m;
		float reg = alpha * L2_regularisation * p[thx];
		//
		p[thx] -= (ch + reg);
	}
};

void moment(
	Mdl_t   * mdl,
	float *** hist,
	uint         i,
	float    alpha,
	uint         t
) {
	FOR(0, i, mdl->insts) {
		Inst_t * inst = mdl->inst[i];
		//
		if (inst->P != 0) {
			kerd_moment<<<dim3(KERD(mdl->inst[i]->P, 256)),dim3(256)>>>(
				t,
				inst->p__d, inst->dp__d,
				hist[0][i],
				alpha,
				inst->P
			);
		}
	}
	ATTENDRE_CUDA();
};