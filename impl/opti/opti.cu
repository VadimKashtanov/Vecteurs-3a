#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../impl_template/tmpl_etc.cu"

uint hists[] = {
	SGD_____HISTOIRE,
	MOMENT__HISTOIRE,
	RMSPROP_HISTOIRE,
	ADAM____HISTOIRE
};

void opti(
	Mdl_t     *     mdl,
	BTCUSDT_t * btcusdt,
	uint      *   ts__d,
	uint              I,
	uint       tous_les,
	uint        methode,
	float         alpha)
{
	//	--- Drop Out ---
	FOR(0, i, mdl->insts) {
		if (mdl->inst[i]->drop_out == 0) {
			mdl->inst[i]->drop_out_oui_non = 0;
		} else {
			uint y_drop_out[mdl->inst[i]->Y];
			FOR(0, j, mdl->inst[i]->Y) y_drop_out[j] = (rnd()<mdl->inst[i]->drop_out ? 1 : 0);
			mdl->inst[i]->drop_out_oui_non = cpu_vers_gpu<uint>(y_drop_out, mdl->inst[i]->Y);
		}
	}

	//	--- Hist ---
	float *** hist = alloc<float**>(hists[methode]);
	FOR(0, h, hists[methode]) {
		hist[h] = alloc<float*>(mdl->insts);
		FOR(0, i, mdl->insts) {
			hist[h][i] = cudalloc<float>(mdl->inst[i]->P);
			// = 0
		}
	}

	//	--- Plume ---
	mdl_plume_grad(mdl, btcusdt, ts__d);
	//
	float _max_abs_grad = 1;//mdl_max_abs_grad(mdl);
	if (_max_abs_grad == 0) ERR("Le grad max est = 0");
	//
	alpha /= _max_abs_grad;
	//
	printf("alpha=%f, max_abs_grad=%f => nouveau alpha=%f\n", alpha, _max_abs_grad, alpha / _max_abs_grad);
	//
	//	--- Opti  ---
	FOR(0, i, I) {
		if (i != 0) {
			//	dF(x)
			mdl_allez_retour(mdl, btcusdt, ts__d);

			//	x = x - dx
			if (methode == SGD    ) sgd    (mdl, hist, i, alpha, i);
			if (methode == MOMENT ) moment (mdl, hist, i, alpha, i);
			if (methode == RMSPROP) rmsprop(mdl, hist, i, alpha, i);
			if (methode == ADAM   ) adam   (mdl, hist, i, alpha, i);
		}
		//
		if (i % tous_les == 0) {
			float s = mdl_S(mdl, btcusdt, ts__d);
			float * p0 = mdl_pourcent(mdl, btcusdt, ts__d, 0.0);
			float * p1 = mdl_pourcent(mdl, btcusdt, ts__d, 1.0);
			float * p8 = mdl_pourcent(mdl, btcusdt, ts__d, 4.0);
			//

			printf("%3.i/%3.i score = %f (", i, I, s);

			printf("^0:{");
			FOR(0, p, btcusdt->Y) printf("\033[96m%f%%\033[0m ", p0[p]);
			printf("}  ");

			printf("^1:{");
			FOR(0, p, btcusdt->Y) printf("\033[96m%f%%\033[0m ", p1[p]);
			printf("}  ");

			printf("^4:{");
			FOR(0, p, btcusdt->Y) printf("\033[96m%f%%\033[0m ", p8[p]);
			printf("}");

			printf(")\n");

			free(p0);
			free(p1);
			free(p8);
		};
	};
	//
	//
	FOR(0, h, hists[methode]) {
		FOR(0, i, mdl->insts) {
			cudafree<float>(hist[h][i]);
		}
		free(hist[h]);
	}
	free(hist);

	//	--- Drop Out ---
	FOR(0, i, mdl->insts) {
		if (mdl->inst[i]->drop_out == 0) {
			//
		} else {
			cudafree<uint>(mdl->inst[i]->drop_out_oui_non);
			mdl->inst[i]->drop_out_oui_non = 0;
		}
	}
}