#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../impl_template/tmpl_etc.cu"

void mdl_dy_zero(Mdl_t * mdl) {
	FOR(0, i, mdl->insts) {
		uint I = mdl->inst[i]->Y*GRAND_T*MEGA_T;
		kerd_liste_inis<float><<<dim3(KERD(I, 64)), dim3(64)>>>(
			mdl->inst[i]->dy__d, 0.0, I
		);
		if (mdl->inst[i]->P != 0) {
			uint P = mdl->inst[i]->P;
			kerd_liste_inis<float><<<dim3(KERD(P, 64)), dim3(64)>>>(
				mdl->inst[i]->dp__d, 0.0, P
			);
		}
	};
	ATTENDRE_CUDA();
};

static
__global__ void k64(float * l, float * l2, uint I) {
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	//
	uint t = threadIdx.x;
	//
	float __shared__ s[64];
	//64
	float a = (thx*2<I ? l[thx*2] : 0.0);
	float b = (thx*2+1<I ? l[thx*2+1] : 0.0);
	s[t] = MAX2(fabs(a), fabs(b));
	__syncthreads();
	//32
	if (t % 2 == 0) {
		s[t] = MAX2(s[t], s[t+1]);
	}
	__syncthreads();
	//16
	if (t % 4 == 0) {
		s[t] = MAX2(s[t], s[t+1*2]);
	}
	__syncthreads();
	//8
	if (t % 8 == 0) {
		s[t] = MAX2(s[t], s[t+1*2*2]);
	}
	__syncthreads();
	//4
	if (t % 16 == 0) {
		s[t] = MAX2(s[t], s[t+1*2*2*2]);
	}
	__syncthreads();
	//2
	if (t % 32 == 0) {
		s[t] = MAX2(s[t], s[t+1*2*2*2*2]);
	}
	__syncthreads();
	//1
	if (t % 64 == 0) {
		s[t] = MAX2(s[t], s[t+1*2*2*2*2*2]);
	}
	__syncthreads();
	//
	if (t % 64 == 0) {
		l2[thx/64] = s[t];
		//printf(">>%f\n", s[t]);
	}
};

/*static float* div_64(float * l, uint I) {
	float * l2 = cudalloc<float>(I);
	k64<<<dim3(KERD(DIV(I,2),64)), dim3(64)>>>(l, l2, I);
	ATTENDRE_CUDA();
	return l2;
};

static float max_abs_grad_inst(float * l, uint I) {
	if (I != 0) {
		float * l1 = l;
		float nb = logf(I) / logf(64.0);
		FOR(0, i, nb) {
			float * l2 = div_64(l1, I);
			I = KERD(I,64)*64 / 64;
			if (i == 0) {
				l1 = l2;
			} else {
				cudafree<float>(l1);
				l1 = l2;
			}
			if (I == 1) {
				float * _l2 = gpu_vers_cpu<float>(l2, I);
				cudafree<float>(l2);
				float ret = _l2[0];
				free(_l2);
				return ret;
			}
		};
	} else {
		return 0.0;
	}
};*/

float mdl_max_abs_grad(Mdl_t * mdl) {
	float _MAX[mdl->insts];
	uint     I[mdl->insts];
	float   nb[mdl->insts];
	FOR(0, i, mdl->insts) {
		_MAX[i] = 0.0;
		I[i]    = mdl->inst[i]->P;
		nb[i]   = logf(I[i]) / logf(64.0);
	}
	//
	float nb_max = nb[0];
	FOR(0, i, mdl->insts) if (nb_max < nb[i]) nb_max = nb[i];
	//
	float * l1[mdl->insts];
	float * l2[mdl->insts];
	FOR(0, i, mdl->insts) {
		l1[i] = mdl->inst[i]->dp__d;
	}
	//
	FOR(0, _nb, nb_max) {
		FOR(0, i, mdl->insts) {
			if (I[i] > 1) {
				//float * l2 = div_64(l1, I);
				//
				l2[i] = cudalloc<float>(I[i]);
				k64<<<dim3(KERD(DIV(I[i],2), 64)), dim3(64)>>>(l1[i], l2[i], I[i]);
				I[i] = KERD(I[i],64)*64 / 64;
			}
		}
		ATTENDRE_CUDA();
		//
		FOR(0, i, mdl->insts) {
			if (I[i] >= 1) {
				//
				if (_nb == 0) {
					l1[i] = l2[i];
				} else {
					cudafree<float>(l1[i]);
					l1[i] = l2[i];
				}
				if (I[i] == 1) {
					float * _l1 = gpu_vers_cpu<float>(l1[i], I[i]);
					cudafree<float>(l1[i]);
					_MAX[i] = _l1[0];
					//printf("---%f\n", _l1[0]);
					free(_l1);
					I[i] = 0;
				}
			}
		};
	}
	//
	float _max = _MAX[0];
	//
	FOR(1, i, mdl->insts) {
		if (_max < _MAX[i]) _max = _MAX[i];
	}
	//printf(">>>> %f\n", _max);
	//
	return _max;
};