#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../impl_template/tmpl_etc.cu"

void plumer_model  (Mdl_t * mdl) {
	printf(" === Mdl_t : INSTS=%i ===\n", mdl->insts);
	FOR(0, i, mdl->insts) {
		Inst_t * inst = mdl->inst[i];
		uint ID = inst->ID;
		//
		printf("%3.i| ID=%3.i Y=%5.i P=%6.i L=%6.i : ", i, ID, inst->Y, inst->P, inst->L);

		printf("x_Y={"); FOR(0, j, inst_Xs[ID]) printf("%5.i,", inst->x_Y[j]); printf("}, ");
		printf("x_pos={"); FOR(0, j, inst_Xs[ID]) printf("%3.i,", inst->x_pos[j]); printf("}, ");
		printf("x_t={"); FOR(0, j, inst_Xs[ID]) printf("%i,", inst->x_t[j]); printf("}, ");

		printf("params={"); FOR(0, j, inst_PARAMS[ID]) printf("%i,", inst->params[j]); printf("}, ");

		printf(" inst=(%s)\n", inst_Nom[ID]);
	}

	//	Plumer l'ordre pour le shéma optimisé
	printf(" -- Optimisation --\n");
	FOR(0, i, mdl->BLOQUES) {
		printf("b=%i| ", i);
		FOR(0, j, mdl->elements[i]) printf("%i ", mdl->instructions[i][j]);
		printf("\n");
	};
};

void mdl_plume_poid(Mdl_t * mdl) {
	FOR(0, i, mdl->insts) {
		printf("###### %i-mem INST (ID=%i) #######\n", i, mdl->inst[i]->ID);
		float * p = gpu_vers_cpu<float>(mdl->inst[i]->p__d, mdl->inst[i]->P);
		FOR(0, j, mdl->inst[i]->P) printf("%i| %f\n", j, p[j]);
	};
};

void mdl_plume_grad(Mdl_t * mdl, BTCUSDT_t * btcusdt, uint * ts__d) {
	//
	mdl_allez_retour(mdl, btcusdt, ts__d);
	//
	printf("=== Grad ===\n");
	FOR(0, i, mdl->insts) {
		Inst_t * inst = mdl->inst[i];
		if (inst->P != 0) {
			//
			float *  p = gpu_vers_cpu<float>(inst-> p__d, inst->P);
			float * dp = gpu_vers_cpu<float>(inst->dp__d, inst->P);
			//
			//
			float pmax=p[0], pmin=p[0], pabsmax=fabs( p[0]), pmoyabs=fabs( p[0]);
			FOR(1, j, inst->P) {
				float val = p[j];
				if (val > pmax) pmax = val;
				if (val < pmin) pmin = val;
				if (fabs(val) > pabsmax) pabsmax = fabs(val);
				pmoyabs += fabs(val);
			}
			pmoyabs /= (float)inst->P;
			//
			//
			float gmax=dp[0], gmin=dp[0], gabsmax=fabs( dp[0]), gmoyabs=fabs( dp[0]);
			FOR(1, j, inst->P) {
				float val = dp[j];
				if (val > gmax) gmax = val;
				if (val < gmin) gmin = val;
				if (fabs(val) > gabsmax) gabsmax = fabs(val);
				gmoyabs += fabs(val);
			}
			gmoyabs /= (float)inst->P;
			//
			//
			free( p);
			free(dp);
			//
			//
			printf("%3.i| [ID=%2.i] Y=%5.i : Grad max=%+f; min=%+f absmax=%+f moy=%+f ;; Poid max=%+f; min=%+f absmax=%+f moy=%+f ;; Poids=%7.i  (do=%.3g dc=%.3g)\n",
				i, inst->ID, inst->Y,
				gmax, gmin, gabsmax, gmoyabs,
				pmax, pmin, pabsmax, pmoyabs,
				inst->P,
				inst->drop_out,
				inst->drop_connect
			);
		};
	};
};