#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../impl_template/tmpl_etc.cu"

__global__
static void kerd_p1e5(float * p__d, uint p, float _1E5) {
	p__d[p] += _1E5;
};

static void plus_1e5(float * p__d, uint p, float _1E5) {
	kerd_p1e5<<<1,1>>>(p__d, p, _1E5);
	ATTENDRE_CUDA();
};

//	---------------------------------------------------

__global__
static void kerd_lire(float * p__d, uint p, float * val) {
	val[0] = p__d[p];
};

static float lire(float * p__d, uint p) {
	float * val = cudalloc<float>(1);
	kerd_lire<<<1,1>>>(p__d, p, val);
	ATTENDRE_CUDA();
	//
	float * _ret = gpu_vers_cpu<float>(val, 1);
	float ret = _ret[0];
	free(_ret);cudafree<float>(val);
	//
	return ret;
};

void tester_le_model(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	uint ts[GRAND_T];
	FOR(0, t, GRAND_T) ts[t] = rand() % (btcusdt->T - MEGA_T);
	uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
	//
	mdl_verif(mdl, btcusdt);
	//
	//
	mdl_allez_retour(mdl, btcusdt, ts__d);
	//
	//
	INIT_CHRONO(s)
	DEPART_CHRONO(s)
	//
	float S = mdl_S(mdl, btcusdt, ts__d);
	//
	float _1E5 = 5e-3;
	uint lp = 0;
	FOR(0, i, mdl->insts) {
		printf("#### INSTRUCTION %i (%s Y=%i) ####\n",
			i, 
			inst_Nom[mdl->inst[i]->ID], mdl->inst[i]->Y
		);
		//
		float * dp = gpu_vers_cpu<float>(mdl->inst[i]->dp__d, mdl->inst[i]->P);
		//
		FOR(0, p, mdl->inst[i]->P) {

			//	f(x + 1e-5)
			plus_1e5(mdl->inst[i]->p__d, p, _1E5);
			float S1e5 = mdl_S(mdl, btcusdt, ts__d);
			plus_1e5(mdl->inst[i]->p__d, p, -_1E5);

			//	df
			float a = (S1e5 - S)/_1E5;

			//	f'
			float b = dp[p];

			//	vitesse
			float vitesse = (float)(++lp) / VALEUR_CHRONO(s);

			//
			printf("%i| ", p);
			PLUME_CMP(a, b);
			if (b != 0) printf(" (x%+f) ", a/b);
			printf(" (%+f m/s)   inst=%i\n", vitesse, i);
		};
		free(dp);
	};
	printf("1E5  === dp\n");
	//
	cudafree<uint>(ts__d);
};