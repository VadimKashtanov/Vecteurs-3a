#include "hip/hip_runtime.h"
#include "union.cuh"

__global__
static void kerd__union(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = (_y<X0 ? x0[tx0*X0 + _y] : x1[tx1*X1 + _y - X0]);
	};
};

__global__
static void kerd__union__x0existe(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		//uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = (_y<X0 ? x0[tx0*X0 + _y] : 0.0);
	};
};

__global__
static void kerd__union__x1existe(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		//uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = (_y<X0 ? 0.0 : x1[tx1*X1 + _y - X0]);
	};
};

void union__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	if (x0_existe && x1_existe) {
		kerd__union<<<dim3(KERD(Y,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t
		);
	} else if (x0_existe) {
		kerd__union__x0existe<<<dim3(KERD(Y,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t
		);
	} else if (x1_existe) {
		kerd__union__x1existe<<<dim3(KERD(Y,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};