#include "hip/hip_runtime.h"
#include "imaxmin.cuh"

#include "../impl_template/tmpl_etc.cu"

/*static __global__ void d_kerd_imaxmin(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y, uint L,
	float * y, float * l,
	float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	//uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint thx = threadIdx.x;
	uint _c0 = blockIdx.x;
	//
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint _X = X0 / C0;
		//
		float dmax = dy[ty*2*C0 + 2*_c0 + 0];
		float dmin = dy[ty*2*C0 + 2*_c0 + 1];
		uint omax = (uint)l[ty*2*C0 + 2*_c0 + 0];
		uint omin = (uint)l[ty*2*C0 + 2*_c0 + 1];
		//
		atomicAdd(&dx0[tx0*X0 + _c0*_X + omax], dmax);
		atomicAdd(&dx0[tx0*X0 + _c0*_X + omin], dmin);
	};
}*/

static __global__ void d_kerd_imaxmin(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y, //uint    L,
	float * y, //float * l,
	float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint _X  = X0 / C0;
		uint _c0 = (_x - (_x%_X)) / _X;
		//
		float __x = x0[tx0*X0 + _x ];
		if (y[ty*Y + _c0*2+0] == __x) {	//max
			atomicAdd(&dx0[tx0*X0 + _x ], dy[ty*Y + _c0*2+0]);
		}
		if (y[ty*Y + _c0*2+1] == __x) {	//min
			atomicAdd(&dx0[tx0*X0 + _x ], dy[ty*Y + _c0*2+1]);
		}
	}
}

void imaxmin__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint \
		C0     = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	if (x0_existe) {
		/*d_kerd_imaxmin<<<dim3(C0, KERD(GRAND_T,1)), dim3(1024,1)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,   inst->L,
			inst->y__d, inst->l__d,
			inst->dy__d,
			//
			mega_t,
			//
			C0
		);*/
		d_kerd_imaxmin<<<dim3(KERD(inst->x_Y[0]*C0,16), KERD(GRAND_T,16)), dim3(16,16)>>>(	// faire Min ET Max !!!
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,//  inst->L,
			inst->y__d,// inst->l__d,
			inst->dy__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		// rien
	}
};