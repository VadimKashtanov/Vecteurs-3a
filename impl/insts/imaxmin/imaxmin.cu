#include "hip/hip_runtime.h"
#include "imaxmin.cuh"

#include "../impl_template/tmpl_etc.cu"

uint imaxmin__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

uint imaxmin__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint C0     = params[0];
	return 0;//2 * C0; //max, min
};

void imaxmin__init_poids(Inst_t * inst) {
	uint C0     = inst->params[0];
	//
	ASSERT(inst->Y == 2*C0);
	ASSERT(C0 > 0);
};

static __global__ void poser_FLT_MAX(
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = (_y%2==0 ? -FLT_MAX : FLT_MAX);	//max , min
	}
}

void imaxmin__pre_f(Inst_t * inst) {
	uint C0 = inst->params[0];
	FOR(0, mega_t, MEGA_T) {
		poser_FLT_MAX<<<dim3(KERD(inst->Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	}
};