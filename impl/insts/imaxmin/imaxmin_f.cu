#include "hip/hip_runtime.h"
#include "imaxmin.cuh"

#include "../impl_template/tmpl_etc.cu"

#define MAX_float 123456789.87654321

/*static __global__ void kerd_imaxmin(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y, uint    L,
	float * y, float * l,
	//
	uint mega_t,
	//
	uint C0)
{
	//uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint thx = threadIdx.x;
	uint _c0 = blockIdx.x;
	//
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		//
		__shared__ float        max[1024];
		__shared__ float        min[1024];
		__shared__ uint origine_max[1024];
		__shared__ uint origine_min[1024];
		//
		uint _X = X0 / C0;
		//
		uint _origine = 0*1024 + thx;
		float _val = (_origine<_X ? x0[tx0*X0 + _c0*_X + _origine] : MAX_float);
		//
		float _max = _val;
		float _min = _val;
		uint _origine_max = _origine;
		uint _origine_min = _origine;
		//
		FOR(1, i, KERD(_X, 1024)) {
			uint p = i*1024+thx;
			if (p < _X) {
				_val = x0[tx0*X0 + _c0*_X + p];
				//
				if (_val > _max) {
					_max = _val;
					_origine_max = p;
				};
				if (_val < _min) {
					_min = _val;
					_origine_min = p;
				};
			};
		}
		max[thx] = _max;
		min[thx] = _min;
		origine_max[thx] = _origine_max;
		origine_min[thx] = _origine_min;
		//
		uint thx_a, thx_b;
		float max_a, max_b, min_a, min_b;
		uint omax_a, omax_b, omin_a, omin_b;
		__syncthreads();
		//
		FOR(0, i, 10) {
			uint modulo = pow(2, 1 + i);
			if (!(modulo >= 2 && modulo <= 1024)) {
				printf("modulo = %i\n", modulo);
				assert(0);
			}
			if (thx % modulo == 0) {
				//
				thx_a = thx +   0    ;
				thx_b = thx +modulo-1;
				//
				max_a  = max        [thx_a]; max_b  = max        [thx_b];
				omax_a = origine_max[thx_a]; omax_b = origine_max[thx_b];
				if (max_a == MAX_float) {
					max[thx] = max_b;
					origine_max[thx] = omax_b;
				} else if (max_b == MAX_float) {
					max[thx] = max_a;
					origine_max[thx] = omax_a;
				} else if (max_a > max_b) {
					max[thx] = max_b;
					origine_max[thx] = omax_b;
				} else {
					max[thx] = max_a;
					origine_max[thx] = omax_a;
				}
				//
				min_a  = min        [thx_a]; min_b  = min        [thx_b];
				omin_a = origine_min[thx_a]; omin_b = origine_min[thx_b];
				if (min_a == MAX_float) {
					min[thx] = min_b;
					origine_min[thx] = omin_b;
				} else if (min_b == MAX_float) {
					min[thx] = min_a;
					origine_min[thx] = omin_a;
				} else if (min_a > min_b) {
					min[thx] = min_b;
					origine_min[thx] = omin_b;
				} else {
					min[thx] = min_a;
					origine_min[thx] = omin_a;
				}
				//
			}
			__syncthreads();
		}
		//
		y[ty*2*C0 + 2*_c0 + 0] = max[0];
		y[ty*2*C0 + 2*_c0 + 1] = min[0];
		l[ty*2*C0 + 2*_c0 + 0] = (float)origine_max[0];
		l[ty*2*C0 + 2*_c0 + 1] = (float)origine_min[0];
		printf("_c0=%i %f %f  (%f %f %f %f)\n", _c0, max[0], min[0],   max[0],max[1],max[2],max[3]);
	};
}*/

static __global__ void kerd_imaxmin(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,// uint    L,
	float * y,// float * l,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint _X  = X0 / C0;
		uint _c0 = (_x - (_x%_X)) / _X;
		//
		float __x = x0[tx0*X0 + _x ];
		atomicMax(&y[ty*Y + _c0*2+0], __x);
		atomicMin(&y[ty*Y + _c0*2+1], __x);
	}
}

void imaxmin__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint \
		C0     = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	inst_zero_mega_t(inst, mega_t);
	//
	if (x0_existe) {
		/*kerd_imaxmin<<<dim3(C0, KERD(GRAND_T,1)), dim3(1024,1)>>>(	// faire Min ET Max !!!
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,  inst->L,
			inst->y__d, inst->l__d,
			//
			mega_t,
			//
			C0
		);*/
		kerd_imaxmin<<<dim3(KERD(inst->x_Y[0]*C0,16), KERD(GRAND_T,16)), dim3(16,16)>>>(	// faire Min ET Max !!!
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,//  inst->L,
			inst->y__d,// inst->l__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};