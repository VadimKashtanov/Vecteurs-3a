#include "hip/hip_runtime.h"
#include "QKtDivClef.cuh"

static __global__ void kerd__QKtDivClef__simple(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ay*C0
	uint _ay = thx % Ay;
	uint _c0 = (thx-_ay)/Ay;

	//	thy = Bx*GRAND_T
	uint _bx = thy % Bx;
	uint  _t = (thy-_bx)/Bx;

	if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		uint pos_y = ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx;
		FOR(0, k, Ax) {
			uint pos_x0 = tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _ay*Ax + k;
			//uint pos_x1 = tx1*C0*Bx*Ax + _c0*(Bx*Ax) + k*Bx + _bx;
			//
			uint pos_x1_transpose = tx1*C0*Bx*Ax + _c0*(Bx*Ax) + _bx*Ax + k;
			//
			s += x0[pos_x0] * x1[pos_x1_transpose];
		}
		y[pos_y] = s / sqrtf((float)Ax);
	}
};

//	---------------------------------------------------------------------------------

void QKtDivClef__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	if (x0_existe && x1_existe) {
		kerd__QKtDivClef__simple<<<dim3(KERD((Ay*C0),16), KERD((Bx*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, Bx, C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};