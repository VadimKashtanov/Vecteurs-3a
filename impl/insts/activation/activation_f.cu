#include "hip/hip_runtime.h"
#include "activation.cuh"

__global__
static void kerd__activation(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y, uint    L,
	float * y, float * l,
	//
	uint mega_t,
	//
	uint activ)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = x0[tx0*X0 + _y];
		//
		float a = ACTIVATION(activ, s);
		//
		y[ty*Y + _y] = a;
		l[ty*L + _y] = s;
	};
};

void activation__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__activation<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y, inst->L,
			inst->y__d, inst->l__d,
			//
			mega_t,
			//
			inst->params[0]
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};