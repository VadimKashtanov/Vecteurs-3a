#include "hip/hip_runtime.h"
#include "activation.cuh"

__global__
static void d_kerd__activation(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y, uint    L,
	float * y, float * l,
	float * dy,
	//
	uint mega_t,
	//
	uint activ)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );

		float a = y[ty*Y + _y];
		float s = l[ty*L + _y];

		float da = d_ACTIVATION(activ, s, a);

		atomicAdd(&dx0[tx0*X0 + _y], dy[ty*Y + _y] * da);
	};
};

void activation__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__activation<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y, inst->L,
			inst->y__d, inst->l__d,
			inst->dy__d,
			//
			mega_t,
			//
			inst->params[0]
		);
	} else {
		// rien
	}
};