#include "hip/hip_runtime.h"
#include "matmul_poid_AP.cuh"

static __global__ void kerd__matmul_poid_AP__simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ay*C0
	uint _ay = thx % Ay;
	uint _c0 = (thx-_ay)/Ay;

	//	thy = Bx*GRAND_T
	uint _bx = thy % Bx;
	uint  _t = (thy-_bx)/Bx;

	if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		uint pos_y = ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx;
		FOR(0, k, Ax) {
			uint pos_x0 = tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _ay*Ax + k;
			uint pos_p  =                _c0*(Bx*Ax) + k*Bx + _bx;
			//
			s += x0[pos_x0] * p[pos_p];
		}
		y[pos_y] = s;
	}
};

//	---------------------------------------------------------------------------------

void matmul_poid_AP__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	//ASSERT(x0_existe);
	//
	if (x0_existe) {
		kerd__matmul_poid_AP__simple<<<dim3(KERD((Ay*C0),16), KERD((Bx*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->p__d,
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, Bx, C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};