#include "hip/hip_runtime.h"
#include "div_scal.cuh"

static __global__ void d_kerd_div_scal(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		//y[ty*Y + _y] = x0[tx0*X0 + _y] / x1[tx1*X1 + 0];
		uint c0 = (  _y - (_y%(X0/C0))  )/(X0/C0);
		//
		float a = x0[tx0*X0 + _y];
		float b = x1[tx1*X1 + c0];
		//
		float _dy = dy[ty*Y + _y];
		//
		//printf("%f a=%f b=%f %f %f\n", _dy, a, b, _dy / b, _dy * (-1) * a/(b*b));
		//
		atomicAdd(&dx0[tx0*X0 + _y], _dy / b);				// ils s'annulent entre
		atomicAdd(&dx1[tx1*X1 + c0], _dy * (-1) * a/(b*b)); // eux memes (indirectement)
	};
}

void div_scal__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	//
	if (x0_existe && x1_existe) {
		d_kerd_div_scal<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};