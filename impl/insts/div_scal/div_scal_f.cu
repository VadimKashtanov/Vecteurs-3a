#include "hip/hip_runtime.h"
#include "div_scal.cuh"

static __global__ void kerd_div_scal(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint c0 = (  _y - (_y%(X0/C0))  )/(X0/C0);
		//
		y[ty*Y + _y] = x0[tx0*X0 + _y] / x1[tx1*X1 + c0];
		//printf("%f\n", x1[tx1*X1 + C0]);
	};
}

void div_scal__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	//inst_zero_mega_t(inst, mega_t);
	//
	if (x0_existe && x1_existe) {
		kerd_div_scal<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};