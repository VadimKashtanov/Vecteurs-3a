#include "activation_poid.cuh"

uint activation_poid__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return Y;
};

uint activation_poid__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return Y;
};

void activation_poid__init_poids(Inst_t * inst) {
	float p[inst->P];
	FOR(0, i, inst->P) p[i] = poid_1_1()*0 * 0.5;

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};

void activation_poid__pre_f(Inst_t * inst) {
	
};