#include "hip/hip_runtime.h"
#include "sous.cuh"

__global__
static void kerd__sub1(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	float sng_soustraction)
{
	//
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = sng_soustraction*x0[tx0*X0 + _y];
	}
};

__global__
static void kerd__sub2(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t)
{
	//
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = x0[tx0*X0 + _y] - x1[tx1*X1 + _y];
	}
};

void sous__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint X1 = inst->x_Y[1];	uint x1_t = inst->x_t[1];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	uint xs_existants = x0_existe + x1_existe;
	//
	if (xs_existants == 2) {
		kerd__sub2<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			x0_t, X0, x__d[0],
			x1_t, X1, x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t
		);
	} else if (xs_existants == 1) {
		uint _i0 = (x0_existe ? 0 : 1);
		//
		float sng_soustraction = (_i0 == 0 ? +1 : -1);
		//
		kerd__sub1<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[_i0], inst->x_Y[_i0], x__d[_i0],
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			//
			sng_soustraction
		);
	} else if (xs_existants == 0) {
		inst_zero_mega_t(inst, mega_t);
	}
};