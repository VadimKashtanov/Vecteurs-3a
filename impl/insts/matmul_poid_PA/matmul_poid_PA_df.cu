#include "hip/hip_runtime.h"
#include "matmul_poid_PA.cuh"

static __global__ void d_kerd__matmul_poid_PA__simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ay*C0
	uint _ay = thx % Ay;
	uint _c0 = (thx-_ay)/Ay;

	//	thy = Bx*GRAND_T
	uint _bx = thy % Bx;
	uint  _t = (thy-_bx)/Bx;

	if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint pos_y = ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx;
		float _dy = dy[pos_y];
		//
		FOR(0, k, Ax) {
			uint pos_p  =                _c0*(Ax*Ay) + _ay*Ax + k;
			uint pos_x0 = tx0*C0*Bx*Ax + _c0*(Bx*Ax) + k*Bx + _bx;
			//
			//s += x0[pos_x0] * x1[pos_x0];
			atomicAdd(&dx0[pos_x0], p [pos_p ] * _dy);
			atomicAdd(&dp [pos_p ], x0[pos_x0] * _dy);
		}
	}
};

//	---------------------------------------------------------------------------------

void matmul_poid_PA__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	//ASSERT(x0_existe);
	//
	if (x0_existe) {
		d_kerd__matmul_poid_PA__simple<<<dim3(KERD((Ay*C0),16), KERD((Bx*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->p__d, inst->dp__d,
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, Bx, C0
		);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};