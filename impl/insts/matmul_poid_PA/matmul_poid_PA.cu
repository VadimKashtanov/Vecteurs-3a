#include "hip/hip_runtime.h"
#include "matmul_poid_PA.cuh"

uint matmul_poid_PA__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	return C0 * Ax*Ay;//C0 * Bx*Ax;
};

uint matmul_poid_PA__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void matmul_poid_PA__init_poids(Inst_t * inst) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	ASSERT(inst->Y      == C0 * Bx*Ay);
	//
	ASSERT(inst->x_Y[0] == C0 * Bx*Ax);

	float p[inst->P];
	FOR(0, i, inst->P) p[i] = poid_1_1() * sqrtf(6.0/(float)Ax);

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};

void matmul_poid_PA__pre_f(Inst_t * inst) {
	
};