#include "hip/hip_runtime.h"
#include "div.cuh"

__global__
static void kerd__div(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t)
{
	//
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		y[ty*Y + _y] = x0[tx0*X0 + _y] / x1[tx1*X1 + _y];
	}
};

void div__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0];	uint x0_t = inst->x_t[0];
	uint X1 = inst->x_Y[1];	uint x1_t = inst->x_t[1];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	uint xs_existants = x0_existe + x1_existe;
	//
	if (xs_existants == 2) {
		kerd__div<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			x0_t, X0, x__d[0],
			x1_t, X1, x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};