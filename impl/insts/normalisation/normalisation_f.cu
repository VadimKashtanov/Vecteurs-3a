#include "hip/hip_runtime.h"
#include "normalisation.cuh"

__global__
static void kerd__normalisation(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint _c0 = (_x - (_x % (X0/C0)))/(X0/C0);
		//
		float _x0 = x0[tx0*X0 + _x];
		float _max = x1[tx1*2*C0 + _c0*2 + 0];
		float _min = x1[tx1*2*C0 + _c0*2 + 1];
		//
		assert(_max != _min);
		//
		y[ty*Y + _x] = (_x0-_min)/(_max-_min);
	};
};

void normalisation__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__normalisation<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			inst->params[0]
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};