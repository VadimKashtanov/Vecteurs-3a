#include "hip/hip_runtime.h"
#include "activation.cuh"

__global__
static void d_kerd__normalisation(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y,
	float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint _c0 = (_x - (_x % (X0/C0)))/(X0/C0);
		//
		float _x0 = x0[tx0*X0 + _x];
		float _max = x1[tx1*2*C0 + _c0*2 + 0];
		float _min = x1[tx1*2*C0 + _c0*2 + 1];
		//
		//y[ty*Y + _x] = (_x0-_min)/(_max-_min);
		//l[ty*L + _x] = 1/(_max-_min);
		//
		float _dy = dy[ty*Y + _x];
		//
		float _dx0 = _dy / (_max - _min);
		float _dmin = _dy * (-1/(_max-_min) + (-1)*(-1)*(_x0-_min)/powf(_max-_min,2) );
		float _dmax = _dy * (  -(_x0-_min)/powf(_max-_min,2));
		//
		atomicAdd(&dx0[tx0*X0 + _x], _dx0);
		atomicAdd(&dx1[tx1*2*C0 + _c0*2 + 0], _dmax);
		atomicAdd(&dx1[tx1*2*C0 + _c0*2 + 1], _dmin);
	};
};

void normalisation__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__normalisation<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
			//
			inst->Y,
			inst->y__d,
			inst->dy__d,
			//
			mega_t,
			//
			inst->params[0]
		);
	} else {
		// rien
	}
};