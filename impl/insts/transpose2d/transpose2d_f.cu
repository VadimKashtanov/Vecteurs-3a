#include "hip/hip_runtime.h"
#include "transpose2d.cuh"

static __global__ void kerd__transpose2d__simple(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ax*Ay
	uint _ax = thx % Ax;
	uint _ay = (thx-_ax)/Ax;

	//	A faire :
	//	1) Regler la Transpose2d
	//	2) Ajouter des Scaled ( 1/sqrt(Dk) ) pour reajuster le attention mechanisme
	//	3) Ajouter des Masques au Attention (ou pas)
	//	4) Eventuellement jouer avec le softmax du attention, ou le normer autrement.
	//	5) Pas besoin de Norme car tous mes vecteurs seront toujours dans [-1;+1] peut importe où


	//	thy = C0*GRAND_T
	uint _c0 = thy % C0;
	uint __t = (thy-_c0)/C0;

	if (_ay < Ay && _ax < Ax && _c0 < C0 && __t < GRAND_T) {
		uint tx0 = t_MODE(__t, mega_t-x0_t);
		uint ty  = t_MODE(__t, mega_t     );
		//
		uint A  = tx0*X0 + _c0*(Ax*Ay) + _ay*Ax + _ax;
		uint At = ty*Y   + _c0*(Ax*Ay) + _ax*Ay + _ay;
		//
		//printf("%i %i %i %i\n", _ax, _ay, _c0, __t);
		//
		y[At] = x0[A];
	}
};

//	---------------------------------------------------------------------------------

void transpose2d__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		C0 =params[2];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	if (x0_existe) {
		kerd__transpose2d__simple<<<dim3(KERD((Ax*Ay),16), KERD((C0*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};