#include "hip/hip_runtime.h"
#include "transpose2d.cuh"

static __global__ void d_kerd__transpose2d__simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ax*Ay
	uint _ax = thx % Ax;
	uint _ay = (thx-_ax)/Ax;

	//	thy = C0*GRAND_T
	uint _c0 = thy % C0;
	uint __t = (thy-_c0)/C0;

	if (_ay < Ay && _ax < Ax && _c0 < C0 && __t < GRAND_T) {
		uint tx0 = t_MODE(__t, mega_t-x0_t);
		uint ty  = t_MODE(__t, mega_t     );
		//
		uint A  = tx0*X0 + _c0*(Ax*Ay) + _ay*Ax + _ax;
		uint At = ty*Y   + _c0*(Ax*Ay) + _ax*Ay + _ay;
		//
		//y[At] = x0[A];
		atomicAdd(&x0[A], y[At]);
	}
};

//	---------------------------------------------------------------------------------

void transpose2d__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		C0 =params[2];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	if (x0_existe) {
		d_kerd__transpose2d__simple<<<dim3(KERD((Ax*Ay),16), KERD((C0*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, C0
		);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};