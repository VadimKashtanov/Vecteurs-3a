#include "hip/hip_runtime.h"
#include "entree.cuh"

__global__
static void kerd__entree(
	uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint ts = ts__d[_t] + mega_t;
		uint ty = t_MODE(_t, mega_t);
		//
		y[ty*Y + _y] = x0[ts*X0 + _y];
	};
};

void entree__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint Y = inst->Y;
	//
	kerd__entree<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
		inst->x_Y[0], x__d[0],
		//
		inst->Y,
		inst->y__d,
		//
		ts__d, mega_t
	);
};