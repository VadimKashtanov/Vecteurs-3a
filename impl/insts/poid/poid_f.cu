#include "hip/hip_runtime.h"
#include "poid.cuh"

__global__
static void kerd__poid(
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint mega_t)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint ty  = t_MODE(_t, mega_t);
		
		y[ty*Y + _y] = p[_y];
	};
};

void poid__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint Y = inst->Y;
	kerd__poid<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
		inst->p__d,
		//
		inst->Y,
		inst->y__d,
		//
		mega_t
	);
};