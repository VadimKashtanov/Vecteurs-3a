#include "poid.cuh"

uint poid__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return Y;
};

uint poid__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void poid__init_poids(Inst_t * inst) {
	float p[inst->P];
	FOR(0, i, inst->P) p[i] = poid_1_1();

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};

void poid__pre_f(Inst_t * inst) {
	
};