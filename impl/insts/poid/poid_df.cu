#include "hip/hip_runtime.h"
#include "poid.cuh"

__global__
static void d_kerd__poid(
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;
	//
#define _y thx
#define _t thy
	//
	if (_y < Y && _t < GRAND_T) {
		uint ty  = t_MODE(_t, mega_t);
		
		//y[ty*Y + _y] = p[_y];
		atomicAdd(&dp[_y], dy[ty*Y + _y]);
	};
};

void poid__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint Y = inst->Y;
	d_kerd__poid<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
		inst->p__d, inst->dp__d,
		//
		inst->Y,
		inst->y__d, inst->dy__d,
		//
		mega_t
	);
};