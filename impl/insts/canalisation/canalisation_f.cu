#include "hip/hip_runtime.h"
#include "canalisation.cuh"

__global__
static void kerd__canalisation(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float _x = x0[tx0*X0 + _y];
		FOR(0, c0, C0) {
			y[ty*Y + c0*X0 + _y] = _x;
		}
	};
};

void canalisation__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint C0 = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__canalisation<<<dim3(KERD(X0,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};