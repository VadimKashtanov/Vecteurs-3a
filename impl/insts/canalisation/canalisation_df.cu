#include "hip/hip_runtime.h"
#include "canalisation.cuh"

__global__
static void d_kerd__canalisation(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		//float _x = x0[tx0*X0 + _y];
		float d_x = 0;
		FOR(0, c0, C0) {
			d_x += dy[ty*Y + c0*X0 + _y];
		};
		//
		atomicAdd(&dx0[tx0*X0 + _y], d_x);
	};
};

void canalisation__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint C0 = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__canalisation<<<dim3(KERD(X0,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		//	rien
	}
};