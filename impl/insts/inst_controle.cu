#include "hip/hip_runtime.h"
#include "insts.cuh"

__global__
static void kerd_inst_zero_mega_t(float * y, uint Y, uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		y[t_MODE(_t, mega_t)*Y + _y] = 0.0;
	};
};

void inst_zero_mega_t(Inst_t * inst, uint mega_t) {
	//kerd_inst_zero_mega_t<<<DIM2(inst->Y, GRAND_T, 16,16)>>>(
	kerd_inst_zero_mega_t<<<dim3(KERD(inst->Y,16), KERD(GRAND_T,16)),dim3(16,16)>>>(
		inst->y__d,
		inst->Y,
		mega_t
	);
};

//	----------------------------------------------

__global__
static void kerd_drop_out(float * y, uint Y, uint mega_t, uint * oui_non)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		if (oui_non[_y] == 1) {
			y[(t_MODE(_t, mega_t))*Y + _y] = 0.0;
		}
	};
};

void inst_drop_out(Inst_t * inst, uint mega_t) {
	if (inst->drop_out_oui_non != 0) {
		kerd_drop_out<<<dim3(KERD(inst->Y,16), KERD(GRAND_T,16)),dim3(16,16)>>>(
			inst->y__d,
			inst->Y,
			mega_t,
			inst->drop_out_oui_non
		);
	};
};

//	-------------------------------------------------

//	Peut importe la valeur de y (=0 ou pas), il y a un Gradient.
//	Sauf que la on deconnecte un neurone, donc il faut annuler le gradient.

__global__
static void kerd_drop_out_deriv(float * dy, uint Y, uint mega_t, uint * oui_non)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		if (oui_non[_y] == 1) {
			dy[(t_MODE(_t, mega_t))*Y + _y] = 0.0;
		}
	};
};

void inst_drop_out_df(Inst_t * inst, uint mega_t) {
	if (inst->drop_out_oui_non != 0) {
		kerd_drop_out_deriv<<<dim3(KERD(inst->Y,16), KERD(GRAND_T,16)),dim3(16,16)>>>(
			inst->dy__d,
			inst->Y,
			mega_t,
			inst->drop_out_oui_non
		);
	};
};