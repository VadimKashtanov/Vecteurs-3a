#include "hip/hip_runtime.h"
#include "isomme.cuh"

static __global__ void kerd_isomme(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = x0[tx0*X0 + _x];
		//
		uint c0 = (  _x - (_x%(X0/C0))  )/(X0/C0);
		//printf("%i\n", c0);
		//
		atomicAdd(&y[ty*Y + c0], s);
	};
}

void isomme__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	inst_zero_mega_t(inst, mega_t);
	//
	if (x0_existe) {
		kerd_isomme<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};