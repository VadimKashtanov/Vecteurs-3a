#include "hip/hip_runtime.h"
#include "isomme.cuh"

static __global__ void d_kerd_isomme(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y,
	float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint c0 = (  _x - (_x%(X0/C0))  )/(X0/C0);
		//
		atomicAdd(&dx0[tx0*X0 + _x], dy[ty*Y + c0]);
	};
}

void isomme__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	if (x0_existe) {
		d_kerd_isomme<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d,
			inst->dy__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		// rien
	}
};