#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_template/tmpl_etc.cu"

__global__
static void kerd_lire(float * p__d, uint p, float * val) {
	val[0] = p__d[p];
};

static float lire(float * p__d, uint p) {
	float * val = cudalloc<float>(1);
	kerd_lire<<<1,1>>>(p__d, p, val);
	ATTENDRE_CUDA();
	//
	float * _ret = gpu_vers_cpu<float>(val, 1);
	float ret = _ret[0];
	free(_ret);cudafree<float>(val);
	//
	return ret;
};

static float ** toutes_les_predictions(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	ASSERT(btcusdt->T % MEGA_T == 0);
	//
	uint T     = (btcusdt->T - (btcusdt->T % MEGA_T))/MEGA_T;
	uint PREDS = T * MEGA_T;
	//
	float * les_predictions = alloc<float>(PREDS);
	float * les_deltas      = alloc<float>(PREDS);
	
	//
	uint lp = 0;
	//
	FOR(0, _t_, T) {
		//
		uint ts[GRAND_T];
		FOR(0, t, GRAND_T) ts[t] = _t_*MEGA_T + 0;
		//
		uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
		
		//
		mdl_f(mdl, btcusdt, ts__d);
		//
		uint Y = mdl->inst[mdl->inst_sortie]->Y;
		float * y = gpu_vers_cpu<float>(mdl->inst[mdl->inst_sortie]->y__d, GRAND_T*MEGA_T*Y);
		FOR(0, mega_t, MEGA_T) {
			uint ty = t_MODE(0, mega_t);
			les_predictions[lp] = y[ty*Y + 0];
			les_deltas     [lp] = lire(btcusdt->sorties__d, (ts[0] + mega_t)*btcusdt->Y+0);
			lp++;
		}

		//
		cudafree<uint>(ts__d);
		free(y);
	};
	//
	float ** ret = alloc<float*>(2);
	ret[0] = les_predictions;
	ret[1] = les_deltas     ;
	return ret;
};

int main() {
	srand(0);
	init_listes_instructions();
	ecrire_structure_generale("structure_generale.bin");
	verif_insts();

	//	=========================================================
	//	=========================================================
	//	=========================================================
	BTCUSDT_t * btcusdt = cree_btcusdt("prixs/tester_model_donnee.bin");

	//	=========================================================
	//	=========================================================
	//	=========================================================

	//	--- Mdl_t ---
	Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	float ** __lp = toutes_les_predictions(mdl, btcusdt);
	float * lp = __lp[0];
	float * dl = __lp[1];

	FILE * fp = FOPEN("les_predictions.bin", "wb");
	//
	uint T     = (btcusdt->T - (btcusdt->T % MEGA_T))/MEGA_T;
	uint PREDS = T * MEGA_T;
	//
	FWRITE(lp, sizeof(float), PREDS, fp);	//les prédictions
	free(lp);
	//
	FWRITE(dl, sizeof(float), PREDS, fp);	//les déltas
	free(dl);
	//
	fclose(fp);

	//	=========================================================
	//	=========================================================
	//	=========================================================
	//
	//plumer_le_score(mdl, btcusdt);

	//
	liberer_mdl    (mdl    );
	liberer_btcusdt(btcusdt);
};